#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils.cuh"

__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        float value = 0;
        for (int k = 0; k < N; ++k)
        {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

void matrixMultiply(float *h_A, float *h_B, float *h_C, int N)
{
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matrixMultiplyKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()
{
    int N = 1024;
    size_t size = N * N * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < N * N; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    matrixMultiply(h_A, h_B, h_C, N);

    printf("Matrix multiplication completed successfully\n");

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}