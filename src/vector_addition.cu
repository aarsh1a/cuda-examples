#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"

__global__ void vectorAdd(const float *A, const float *B, float *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

void addVectors(const float *h_A, const float *h_B, float *h_C, int N)
{
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    vectorAdd<<<numBlocks, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()
{
    int N = 1048576;
    size_t size = N * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < N; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    addVectors(h_A, h_B, h_C, N);

    printf("Vector addition completed successfully\n");

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}